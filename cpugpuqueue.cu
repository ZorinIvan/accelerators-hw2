/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <unistd.h>
#include <assert.h>
#include <string.h>

#define IMG_DIMENSION 32
#define N_IMG_PAIRS 10000
#define NREQUESTS 10000
#define N_STREMS 64
#define HIST_SIZE 256

typedef unsigned char uchar;
#define OUT

#define CUDA_CHECK(f) do {                                                                  \
    hipError_t e = f;                                                                      \
    if (e != hipSuccess) {                                                                 \
        printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));    \
        exit(1);                                                                            \
    }                                                                                       \
} while (0)

#define SQR(a) ((a) * (a))

#define QUEUE_SIZE 10


__device__ __host__ bool is_in_image_bounds(int i, int j) {
    return (i >= 0) && (i < IMG_DIMENSION) && (j >= 0) && (j < IMG_DIMENSION);
}

__device__ __host__ uchar local_binary_pattern(uchar *image, int i, int j) {
    uchar center = image[i * IMG_DIMENSION + j];
    uchar pattern = 0;
    if (is_in_image_bounds(i - 1, j - 1)) pattern |= (image[(i - 1) * IMG_DIMENSION + (j - 1)] >= center) << 7;
    if (is_in_image_bounds(i - 1, j    )) pattern |= (image[(i - 1) * IMG_DIMENSION + (j    )] >= center) << 6;
    if (is_in_image_bounds(i - 1, j + 1)) pattern |= (image[(i - 1) * IMG_DIMENSION + (j + 1)] >= center) << 5;
    if (is_in_image_bounds(i    , j + 1)) pattern |= (image[(i    ) * IMG_DIMENSION + (j + 1)] >= center) << 4;
    if (is_in_image_bounds(i + 1, j + 1)) pattern |= (image[(i + 1) * IMG_DIMENSION + (j + 1)] >= center) << 3;
    if (is_in_image_bounds(i + 1, j    )) pattern |= (image[(i + 1) * IMG_DIMENSION + (j    )] >= center) << 2;
    if (is_in_image_bounds(i + 1, j - 1)) pattern |= (image[(i + 1) * IMG_DIMENSION + (j - 1)] >= center) << 1;
    if (is_in_image_bounds(i    , j - 1)) pattern |= (image[(i    ) * IMG_DIMENSION + (j - 1)] >= center) << 0;
    return pattern;
}

__device__  void gpu_image_to_histogram(uchar *image, int *histogram) {
    uchar pattern = local_binary_pattern(image, threadIdx.x / IMG_DIMENSION, threadIdx.x % IMG_DIMENSION);
    atomicAdd(&histogram[pattern], 1);
}

__device__  void gpu_histogram_distance(int *h1, int *h2, double *distance) {
    int length = 256;
    int tid = threadIdx.x;
    distance[tid] = 0;
    if (h1[tid] + h2[tid] != 0) {
        distance[tid] = ((double)SQR(h1[tid] - h2[tid])) / (h1[tid] + h2[tid]);
    }
    h1[tid] = h2[tid]=0;
    __syncthreads();


    while (length > 1) {
        if (threadIdx.x < length / 2) {
            distance[tid] = distance[tid] + distance[tid + length / 2];
        }
        length /= 2;
        __syncthreads();
    }
}


void image_to_histogram(uchar *image, int *histogram) {
    memset(histogram, 0, sizeof(int) * 256);
    for (int i = 0; i < IMG_DIMENSION; i++) {
        for (int j = 0; j < IMG_DIMENSION; j++) {
            uchar pattern = local_binary_pattern(image, i, j);
            histogram[pattern]++;
        }
    }
}

double histogram_distance(int *h1, int *h2) {
    /* we'll use the chi-square distance */
    double distance = 0;
    for (int i = 0; i < 256; i++) {
        if (h1[i] + h2[i] != 0) {
            distance += ((double)SQR(h1[i] - h2[i])) / (h1[i] + h2[i]);
        }
    }
    return distance;
}

double static inline get_time_msec(void) {
    struct timeval t;
    gettimeofday(&t, NULL);
    return t.tv_sec * 1e+3 + t.tv_usec * 1e-3;
}
/* we won't load actual files. just fill the images with random bytes */
void load_image_pairs(uchar *images1, uchar *images2) {
    srand(0);
    for (int i = 0; i < N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION; i++) {
        images1[i] = rand() % 256;
        images2[i] = rand() % 256;
    }
}


/*************************************************/
/*******CLASS***producer***consumer****queue******/
/*************************************************/
class cpu2gpuQueue {
public:
	cpu2gpuQueue():size(QUEUE_SIZE),head(0),tail(0){/*printf("head=%d\tsize=%d\n",head,size)*/;}
	~cpu2gpuQueue(){}
	__host__ int produce(uchar* imag1,uchar* imag2);
	__device__ int consume(uchar* images);


private:
	volatile int size;
	volatile int head;
	volatile int tail;
	uchar q[QUEUE_SIZE*SQR(IMG_DIMENSION)];
};
__device__ int cpu2gpuQueue::consume(uchar* images)
{
	if(!(tail<head))return 0;
	int i;
	for(i=threadIdx.x;i<2*SQR(IMG_DIMENSION);i+=gridDim.x)
		images[i]=q[(tail%QUEUE_SIZE)*2*SQR(IMG_DIMENSION)+i];
	//make sure all threads copied before increasing the value of tail
	 __syncthreads();
	 if(!threadIdx.x)
	 {
		 size++;
		 tail++;
		 __threadfence_system();
	 }
	// __syncthreads();
	return 1;
}
__host__ int cpu2gpuQueue::produce(uchar* imag1,uchar* imag2)
{
	if(!(head<size)){
		//printf("head=%d\tsize=%d\ttrue\n",head,size);
		return 0;

	}
	memcpy(&q[(head%QUEUE_SIZE)*2*SQR(IMG_DIMENSION)],imag1,SQR(IMG_DIMENSION)*sizeof(uchar));
	memcpy(&q[(head%QUEUE_SIZE)*2*SQR(IMG_DIMENSION)+SQR(IMG_DIMENSION)],imag2,SQR(IMG_DIMENSION)*sizeof(uchar));
	head++;
	return 1;
}

class gpu2cpuQueue {
public:
	gpu2cpuQueue():size(QUEUE_SIZE),head(0),tail(0){}
	~gpu2cpuQueue(){}
	__device__ int produce(double distance);
	__host__ int consume(double* distance);
private:
	volatile int size;
	volatile int head;
	volatile int tail;
	double q[QUEUE_SIZE];
};
__host__ int gpu2cpuQueue::consume(double* distance)
{
	if(!(tail<head))return 0;
	*distance=q[(tail%QUEUE_SIZE)];
	size++;
	tail++;
	return 1;
}
__device__ int gpu2cpuQueue::produce(double distance)
{
	if(!(head<size)) return 0;
	if(threadIdx.x) return 1;
	q[(head%QUEUE_SIZE)]=distance;

	__threadfence_system();
	head++;
	__threadfence_system();
	return 1;
}
struct QP{
	cpu2gpuQueue cpugpu;
	gpu2cpuQueue gpucpu;
};
__global__ void test(struct QP* Ptr){
	int i;
	//if(!threadIdx.x) printf("test kernel\n");
	__shared__ uchar images[2*SQR(IMG_DIMENSION)];
	__shared__ int hist1[HIST_SIZE],hist2[HIST_SIZE];
	__shared__ double distance[SQR(IMG_DIMENSION)];

	if(threadIdx.x<HIST_SIZE)
		hist1[threadIdx.x]=hist2[threadIdx.x]=0;
	i=NREQUESTS;
	while(i--)
	{
		while(!Ptr->cpugpu.consume(images));
		/*if(!threadIdx.x){
			int j;
			for(j=0;j<SQR(IMG_DIMENSION);++j)printf("%d%d",images[+j],images[IMG_DIMENSION * IMG_DIMENSION+j]);
					printf("\n");
		}*/
		gpu_image_to_histogram(images,hist1);
		gpu_image_to_histogram(images+SQR(IMG_DIMENSION),hist2);
		__syncthreads();
		gpu_histogram_distance(hist1,hist2,distance);
		//if(!threadIdx.x) printf("average distance between images %f\n", distance[0] / NREQUESTS);
		__syncthreads();
		while(!Ptr->gpucpu.produce(distance[0]));
	}
}


int main(void) {
	uchar *images1; /* we concatenate all images in one huge array */
	uchar *images2;
	CUDA_CHECK( hipHostAlloc(&images1, N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION, 0) );
	CUDA_CHECK( hipHostAlloc(&images2, N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION, 0) );
	load_image_pairs(images1, images2);
	double t_start, t_finish;
	double total_distance=0,distance=0;
	int i=NREQUESTS,finished=0;
	struct QP *cpuqp,*gpuqp;
	CUDA_CHECK( hipHostAlloc(&cpuqp, sizeof(struct QP), 0) );
	cpuqp->cpugpu=cpu2gpuQueue();
	cpuqp->gpucpu=gpu2cpuQueue();
	CUDA_CHECK( hipHostGetDevicePointer(&gpuqp,cpuqp,0) );


    printf("\n=== CPU ===\n");
    int histogram1[256];
    int histogram2[256];
    t_start  = get_time_msec();
    for (int i = 0; i < NREQUESTS; i++) {
        int img_idx = i % N_IMG_PAIRS;
        image_to_histogram(&images1[img_idx * IMG_DIMENSION * IMG_DIMENSION], histogram1);
        image_to_histogram(&images2[img_idx * IMG_DIMENSION * IMG_DIMENSION], histogram2);
        total_distance += histogram_distance(histogram1, histogram2);
    }
    t_finish = get_time_msec();
    printf("average distance between images %f\n", total_distance / NREQUESTS);
    printf("throughput = %lf (req/sec)\n", NREQUESTS / (t_finish - t_start) * 1e+3);

    total_distance=0;
	test<<<1, 1024>>>(gpuqp);

	printf("after\n");
	i=NREQUESTS;
	while(i--)
	{
		//printf("loop\n");
		distance=0;
		if(cpuqp->gpucpu.consume(&distance))
		{
			total_distance+=distance;
			finished++;

		}
		int img_idx = i % N_IMG_PAIRS,j;
		//for(j=0;j<SQR(IMG_DIMENSION);++j)printf("%d%d",images1[img_idx * IMG_DIMENSION * IMG_DIMENSION+j],images2[img_idx * IMG_DIMENSION * IMG_DIMENSION+j]);
		//printf("\n");
		while(!cpuqp->cpugpu.produce(&images1[img_idx * IMG_DIMENSION * IMG_DIMENSION],&images2[img_idx * IMG_DIMENSION * IMG_DIMENSION]));
	}
	CUDA_CHECK( hipDeviceSynchronize());
	while(finished<NREQUESTS)
	{
		if(cpuqp->gpucpu.consume(&distance))
		{
			total_distance+=distance;
			finished++;

		}
	}
	printf("average distance between images %f\n", total_distance / NREQUESTS);
	return 0;
}
