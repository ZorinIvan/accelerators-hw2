/* compile with: nvcc -O3 -maxrregcount=32 hw2.cu -o hw2 */

#define HW2_CU_

#ifdef HW2_CU_


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <stdlib.h>
#include <unistd.h>
#include <assert.h>
#include <string.h>


//#include "workelement.h"
#define IMG_DIMENSION 32
#define N_IMG_PAIRS 10000
#define NREQUESTS 10000
#define N_STREMS 64
#define HIST_SIZE 256

typedef unsigned char uchar;
#define OUT

#define CUDA_CHECK(f) do {                                                                  \
    hipError_t e = f;                                                                      \
    if (e != hipSuccess) {                                                                 \
        printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));    \
        exit(1);                                                                            \
    }                                                                                       \
} while (0)

#define SQR(a) ((a) * (a))




int calcNumOfThreadblocks(){//TODO: implement
	int ret = 1;
	int nDevices;

	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
	   hipDeviceProp_t prop;
	   hipGetDeviceProperties(&prop, i);
	   
	   int maxThreadsPerBlock = prop.maxThreadsPerBlock;
	   size_t sharedMemPerBlock = prop.sharedMemPerBlock;
	   int regsPerThread = 32;
	   int sm = prop.multiProcessorCount;
	   int maxThreadsPerMultiProcessor = prop.maxThreadsPerMultiProcessor;
	   size_t sharedMemPerMultiprocessor = prop.sharedMemPerMultiprocessor;
	   int regsPerMultiprocessor = prop.regsPerMultiprocessor;
	   
	   //ret += // TODO
	}		
	
	return ret;
			
}



double static inline get_time_msec(void) {
    struct timeval t;
    gettimeofday(&t, NULL);
    return t.tv_sec * 1e+3 + t.tv_usec * 1e-3;
}

/* we'll use these to rate limit the request load */
struct rate_limit_t {
    double last_checked;
    double lambda;
    unsigned seed;
};

void rate_limit_init(struct rate_limit_t *rate_limit, double lambda, int seed) {
    rate_limit->lambda = lambda;
    rate_limit->seed = (seed == -1) ? 0 : seed;
    rate_limit->last_checked = 0;
}

int rate_limit_can_send(struct rate_limit_t *rate_limit) {
    if (rate_limit->lambda == 0) return 1;
    double now = get_time_msec() * 1e-3;
    double dt = now - rate_limit->last_checked;
    double p = dt * rate_limit->lambda;
    rate_limit->last_checked = now;
    if (p > 1) p = 1;
    double r = (double)rand_r(&rate_limit->seed) / RAND_MAX;
    return (p > r);
}

void rate_limit_wait(struct rate_limit_t *rate_limit) {
    while (!rate_limit_can_send(rate_limit)) {
        usleep(1. / (rate_limit->lambda * 1e-6) * 0.01);
    }
}

/* we won't load actual files. just fill the images with random bytes */
void load_image_pairs(uchar *images1, uchar *images2) {
    srand(0);
    for (int i = 0; i < N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION; i++) {
        images1[i] = rand() % 256;
        images2[i] = rand() % 256;
    }
}

__device__ __host__ bool is_in_image_bounds(int i, int j) {
    return (i >= 0) && (i < IMG_DIMENSION) && (j >= 0) && (j < IMG_DIMENSION);
}

__device__ __host__ uchar local_binary_pattern(uchar *image, int i, int j) {
    uchar center = image[i * IMG_DIMENSION + j];
    uchar pattern = 0;
    if (is_in_image_bounds(i - 1, j - 1)) pattern |= (image[(i - 1) * IMG_DIMENSION + (j - 1)] >= center) << 7;
    if (is_in_image_bounds(i - 1, j    )) pattern |= (image[(i - 1) * IMG_DIMENSION + (j    )] >= center) << 6;
    if (is_in_image_bounds(i - 1, j + 1)) pattern |= (image[(i - 1) * IMG_DIMENSION + (j + 1)] >= center) << 5;
    if (is_in_image_bounds(i    , j + 1)) pattern |= (image[(i    ) * IMG_DIMENSION + (j + 1)] >= center) << 4;
    if (is_in_image_bounds(i + 1, j + 1)) pattern |= (image[(i + 1) * IMG_DIMENSION + (j + 1)] >= center) << 3;
    if (is_in_image_bounds(i + 1, j    )) pattern |= (image[(i + 1) * IMG_DIMENSION + (j    )] >= center) << 2;
    if (is_in_image_bounds(i + 1, j - 1)) pattern |= (image[(i + 1) * IMG_DIMENSION + (j - 1)] >= center) << 1;
    if (is_in_image_bounds(i    , j - 1)) pattern |= (image[(i    ) * IMG_DIMENSION + (j - 1)] >= center) << 0;
    return pattern;
}

void image_to_histogram(uchar *image, int *histogram) {
    memset(histogram, 0, sizeof(int) * 256);
    for (int i = 0; i < IMG_DIMENSION; i++) {
        for (int j = 0; j < IMG_DIMENSION; j++) {
            uchar pattern = local_binary_pattern(image, i, j);
            histogram[pattern]++;
        }
    }
}

double histogram_distance(int *h1, int *h2) {
    /* we'll use the chi-square distance */
    double distance = 0;
    for (int i = 0; i < 256; i++) {
        if (h1[i] + h2[i] != 0) {
            distance += ((double)SQR(h1[i] - h2[i])) / (h1[i] + h2[i]);
        }
    }
    return distance;
}

__global__ void gpu_image_to_histogram(uchar *image, int *histogram) {
    uchar pattern = local_binary_pattern(image, threadIdx.x / IMG_DIMENSION, threadIdx.x % IMG_DIMENSION);
    atomicAdd(&histogram[pattern], 1);
}

__device__ void gpu_device_image_to_histogram(uchar *image, int *histogram) {
    uchar pattern = local_binary_pattern(image, threadIdx.x / IMG_DIMENSION, threadIdx.x % IMG_DIMENSION);
    atomicAdd(&histogram[pattern], 1);
}

__global__ void gpu_histogram_distance(int *h1, int *h2, double *distance) {
    int length = 256;
    int tid = threadIdx.x;
    distance[tid] = 0;
    if (h1[tid] + h2[tid] != 0) {
        distance[tid] = ((double)SQR(h1[tid] - h2[tid])) / (h1[tid] + h2[tid]);
    }
    h1[tid] = h2[tid]=0;
    __syncthreads();


    while (length > 1) {
        if (threadIdx.x < length / 2) {
            distance[tid] = distance[tid] + distance[tid + length / 2];
        }
        length /= 2;
        __syncthreads();
    }
}

__device__ void gpu_device_histogram_distance(int *h1, int *h2, double *distance) {

	int length=HIST_SIZE;
	for(int i=threadIdx.x;i<HIST_SIZE;i+=gridDim.x)
	{
	    distance[i] = 0;
	    if (h1[i] + h2[i] != 0) {
	        distance[i] = ((double)SQR(h1[i] - h2[i])) / (h1[i] + h2[i]);
	    }
	    h1[i] = h2[i]=0;
	}
    __syncthreads();

    while (length > 1) {
    	for(int i=threadIdx.x;i<length/2;i+=gridDim.x)
            distance[i] = distance[i] + distance[i + length / 2];
        length /= 2;
        __syncthreads();
    }
}

void print_usage_and_die(char *progname) {
    printf("usage:\n");
    printf("%s streams <load (requests/sec)>\n", progname);
    printf("OR\n");
    printf("%s queue <#threads> <load (requests/sec)>\n", progname);
    exit(1);
}





/*************************************************/
/*******CLASS***WORK***ELEMENT********************/
/*************************************************/




class work_element {
public:
	work_element();
	virtual ~work_element();
	bool is_free(){return free;}
	void do_kernel();
	bool check_kernel_finished();
	void update(uchar* cpu_img1,uchar* cpu_img2,double* start,double* finish){cpu_imgs[0]=cpu_img1;cpu_imgs[1]=cpu_img2;time_start=start;time_finish=finish;}
	static double total_distance;
private:
	hipStream_t stream;
	uchar* cpu_imgs[2];
	uchar* gpu_imgs[2];
	double cpu_distance;
	double * gpu_distance;
	int* gpu_hists[2];
	bool free;
	hipEvent_t events[2];
	double* time_start;
	double* time_finish;
};

double work_element::total_distance=0;






work_element::work_element():cpu_distance(-1),free(true),time_start(NULL),time_finish(NULL) {
	CUDA_CHECK(hipStreamCreate(&stream));
	//CUDA_CHECK(cudaEventCreate(&event));
	CUDA_CHECK(hipMalloc(&gpu_distance,HIST_SIZE*sizeof(double)));
	int i;
	for(i=0;i<2;i++)
	{
		cpu_imgs[i]=NULL;
		CUDA_CHECK(hipMalloc(&gpu_imgs[i],SQR(IMG_DIMENSION)*sizeof(uchar)));
		CUDA_CHECK(hipMalloc(&gpu_hists[i],HIST_SIZE*sizeof(int)));
		CUDA_CHECK(hipEventCreate(&events[i]));
		CUDA_CHECK(hipMemset(gpu_hists[i], 0, HIST_SIZE * sizeof(int)));

	}

}

work_element::~work_element() {
	CUDA_CHECK(hipStreamDestroy(stream));
	CUDA_CHECK(hipFree(gpu_distance));
	int i;
	for(i=0;i<2;i++)
	{
		CUDA_CHECK(hipFree(gpu_imgs[i]));
		CUDA_CHECK(hipFree(gpu_hists[i]));
		CUDA_CHECK(hipEventDestroy(events[i]));
	}

}
void work_element::do_kernel(){
	free=false;
	//*time_finish=get_time_msec();
	//CUDA_CHECK(cudaEventRecord(events[0],stream));
	int i;
	for(i=0;i<2;i++)
	{
		CUDA_CHECK(hipMemcpyAsync(gpu_imgs[i],cpu_imgs[i],SQR(IMG_DIMENSION)*sizeof(uchar),hipMemcpyHostToDevice,stream));
		gpu_image_to_histogram <<< 1, 1024 , 0,stream >>> (gpu_imgs[i], gpu_hists[i]);
	}
	gpu_histogram_distance<<<1, 256>>>(gpu_hists[0], gpu_hists[1], gpu_distance);
	CUDA_CHECK(hipMemcpyAsync(&cpu_distance,gpu_distance,sizeof(double),hipMemcpyDeviceToHost,stream));
	//CUDA_CHECK(cudaEventRecord(events[1],stream));

}

bool work_element::check_kernel_finished(){
	if(free) return false;
	hipError_t ret=hipStreamQuery(stream);
	if(ret==hipErrorInvalidHandle)
	{
		 printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(ret));
		 exit(1);
	}
	if(ret==hipErrorNotReady) return false;
	float tmp=0;
	//CUDA_CHECK(cudaEventElapsedTime(&tmp,events[0],events[1]));
	*time_finish=get_time_msec();;
	total_distance+=cpu_distance;
	free=true;
	return true;


}



/*************************************************/
/*************************************************/




void check_completed_no_block(work_element* streams){
	int i;
	for(i=0;i<N_STREMS;i++)
		streams[i].check_kernel_finished();
}
work_element& find_free_stream(work_element* streams)
{
	int i;
	for(i=0;i<N_STREMS;i++)
	if(streams[i].is_free()) return streams[i];
	while(i||!i)
	{
    	for(i=0;i<N_STREMS;i++)
    		if(streams[i].check_kernel_finished()) return streams[i];
	}
	return streams[1]; //not arrive here
}
void check_completed_block(work_element* streams){
	int i,j=0;
	while(j!=N_STREMS)
	{
    	for(i=0,j=0;i<N_STREMS;i++)
    			j+=(streams[i].is_free()||(int)streams[i].check_kernel_finished());
	}
}

#define QUEUE_SIZE 10
class cpu2gpuQueue {
public:
	cpu2gpuQueue():size(QUEUE_SIZE),head(0),tail(0){}
	~cpu2gpuQueue(){}
	__host__ int produce(int img_idx/*,double* finish*/);
	__device__ int consume(int* img_idx/*,double** finish*/);
private:
	volatile int size;
	volatile int head;
	volatile int tail;
	int q[QUEUE_SIZE];
	/*double* q_finished_times[QUEUE_SIZE];*/
};
__device__ int cpu2gpuQueue::consume(int* img_idx/*,double** finish*/)
{
	if(!(tail<head))return 0;
	*img_idx=q[(tail%QUEUE_SIZE)];
	//*finish=q_finished_times[(tail%QUEUE_SIZE)];
	size++;
	tail++;
	__threadfence_system();
	return 1;
}
__host__ int cpu2gpuQueue::produce(int img_idx/*,double* finish*/)
{
	if(!(head<size))return 0;
	q[(head%QUEUE_SIZE)]=img_idx;
	//q_finished_times[(head%QUEUE_SIZE)]=finish;
	head++;
	return 1;
}
class gpu2cpuQueue {
public:
	gpu2cpuQueue():size(QUEUE_SIZE),head(0),tail(0){}
	~gpu2cpuQueue(){}
	__device__ int produce(double distance/*,double* finish*/);
	__host__ int consume(double* distance);
private:
	volatile int size;
	volatile int head;
	volatile int tail;
	double q[QUEUE_SIZE];
	//double* q_finished_times[QUEUE_SIZE];
};
__host__ int gpu2cpuQueue::consume(double* distance)
{
	if(!(tail<head))return 0;
	*distance=q[(tail%QUEUE_SIZE)];
	//*(q_finished_times[(tail%QUEUE_SIZE)])=get_time_msec();
	size++;
	tail++;
	return 1;
}
__device__ int gpu2cpuQueue::produce(double distance/*,double* finish*/)
{
	if(!(head<size)) return 0;
	q[(head%QUEUE_SIZE)]=distance;
	//q_finished_times[(head%QUEUE_SIZE)]=finish;
	__threadfence_system();
	head++;
	__threadfence_system();
	return 1;
}
typedef struct {
	cpu2gpuQueue cpugpu;
	gpu2cpuQueue gpucpu;
} QP;
typedef QP* QParr;
void checkQueueComplition(int num_of_threadblocks,QP **cpuQPs,int * finished, double* total_distance )
{

	double distance;
	for(int i=0,ret;i<num_of_threadblocks;i++)
	{
		do{
			ret=cpuQPs[i]->gpucpu.consume(&distance);
			*total_distance+=ret*distance;
			*finished+=ret;
		}while(ret);
	}
}
void QueueProduce(int num_of_threadblocks,QP **cpuQPs,int img_idx,int * finished, double* total_distance/*,double* finish_time*/ )
{
	bool produced=false;
	while(!produced)
	{
		for(int i=0;i<num_of_threadblocks;i++)
		{
			if(cpuQPs[i]->cpugpu.produce(img_idx/*,finish_time*/))
			{
				produced=true;
				break;
			}
			else
				checkQueueComplition(num_of_threadblocks,cpuQPs,finished, total_distance );
		}
	}
}
void QueueProduceBlock(int blockId,int num_of_threadblocks,QP **cpuQPs,int img_idx,int * finished, double* total_distance )
{
	bool produced=false;
	while(!produced)
	{
			if(cpuQPs[blockId]->cpugpu.produce(img_idx/*,NULL*/))
			{
				produced=true;
				break;
			}
			else
				checkQueueComplition(num_of_threadblocks,cpuQPs,finished, total_distance );
	}
}

__global__ void kernel_queue_mode(QP** gpuQPs,uchar* imags1,uchar* imags2 ){
	//if(!threadIdx.x) printf("test kernel\n");
	//__shared__ uchar images[2*SQR(IMG_DIMENSION)];
	__shared__ int hist1[HIST_SIZE],hist2[HIST_SIZE];
	__shared__ double distance[HIST_SIZE];
	__shared__ double total_distance;
	__shared__ int img_idx;
	//__shared__ double* finish_time;

	//if(!threadIdx.x) total_distance=0;


	for(int i=threadIdx.x;i<HIST_SIZE;i+=gridDim.x)
		hist1[i]=hist2[i]=0;

	bool running=true;
	//for(int i=0;i<NREQUESTS;i++)
	while( running )
	{
		if(!threadIdx.x)while(!gpuQPs[blockIdx.x]->cpugpu.consume(&img_idx/*,&finish_time*/));
		__syncthreads();
		//printf("img_idx=%d\n",img_idx);
		if(img_idx==-1)
			break;
		gpu_device_image_to_histogram(&imags1[img_idx*SQR(IMG_DIMENSION)],hist1);
		gpu_device_image_to_histogram(&imags2[img_idx*SQR(IMG_DIMENSION)],hist2);
		__syncthreads();
		gpu_device_histogram_distance(hist1,hist2,distance);
		__syncthreads();
		//if(!threadIdx.x)total_distance+=distance[0];
		if(!threadIdx.x)while(!gpuQPs[blockIdx.x]->gpucpu.produce(distance[0]/*,finish_time*/));
		__syncthreads();
	}
	__syncthreads();
	//if(!threadIdx.x)printf("gpu average distance between images %f\n", total_distance / NREQUESTS);
}






















enum {PROGRAM_MODE_STREAMS = 0, PROGRAM_MODE_QUEUE};
int main(int argc, char *argv[]) {

    int mode = -1;
    int threads_queue_mode = -1; /* valid only when mode = queue */
    double load = 0;
    if (argc < 3) print_usage_and_die(argv[0]);

    if        (!strcmp(argv[1], "streams")) {
        if (argc != 3) print_usage_and_die(argv[0]);
        mode = PROGRAM_MODE_STREAMS;
        load = atof(argv[2]);
    } else if (!strcmp(argv[1], "queue")) {
        if (argc != 4) print_usage_and_die(argv[0]);
        mode = PROGRAM_MODE_QUEUE;
        threads_queue_mode = atoi(argv[2]);
        load = atof(argv[3]);
    } else {
        print_usage_and_die(argv[0]);
    }

    uchar *images1; /* we concatenate all images in one huge array */
    uchar *images2;
    CUDA_CHECK( hipHostAlloc(&images1, N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION, 0) );
    CUDA_CHECK( hipHostAlloc(&images2, N_IMG_PAIRS * IMG_DIMENSION * IMG_DIMENSION, 0) );

    load_image_pairs(images1, images2);
    double t_start, t_finish;
    double total_distance;

    /* using CPU */
    printf("\n=== CPU ===\n");
    int histogram1[256];
    int histogram2[256];
    t_start  = get_time_msec();
    for (int i = 0; i < NREQUESTS; i++) {
        int img_idx = i % N_IMG_PAIRS;
        image_to_histogram(&images1[img_idx * IMG_DIMENSION * IMG_DIMENSION], histogram1);
        image_to_histogram(&images2[img_idx * IMG_DIMENSION * IMG_DIMENSION], histogram2);
        total_distance += histogram_distance(histogram1, histogram2);
    }
    t_finish = get_time_msec();
    printf("average distance between images %f\n", total_distance / NREQUESTS);
    printf("throughput = %lf (req/sec)\n", NREQUESTS / (t_finish - t_start) * 1e+3);

    /* using GPU task-serial.. just to verify the GPU code makes sense */
    printf("\n=== GPU Task Serial ===\n");
    do {
        uchar *gpu_image1, *gpu_image2; // TODO: allocate with cudaMalloc
        int *gpu_hist1, *gpu_hist2; // TODO: allocate with cudaMalloc
        double *gpu_hist_distance; //TODO: allocate with cudaMalloc
        double cpu_hist_distance;
        hipMalloc(&gpu_image1, IMG_DIMENSION * IMG_DIMENSION);
        hipMalloc(&gpu_image2, IMG_DIMENSION * IMG_DIMENSION);
        hipMalloc(&gpu_hist1, 256 * sizeof(int));
        hipMalloc(&gpu_hist2, 256 * sizeof(int));
        hipMalloc(&gpu_hist_distance, 256 * sizeof(double));

        total_distance = 0;
        t_start = get_time_msec();
        for (int i = 0; i < NREQUESTS; i++) {
            int img_idx = i % N_IMG_PAIRS;
            hipMemcpy(gpu_image1, &images1[img_idx * IMG_DIMENSION * IMG_DIMENSION], IMG_DIMENSION * IMG_DIMENSION, hipMemcpyHostToDevice);
            hipMemcpy(gpu_image2, &images2[img_idx * IMG_DIMENSION * IMG_DIMENSION], IMG_DIMENSION * IMG_DIMENSION, hipMemcpyHostToDevice);
            hipMemset(gpu_hist1, 0, 256 * sizeof(int));
            hipMemset(gpu_hist2, 0, 256 * sizeof(int));
            gpu_image_to_histogram<<<1, 1024>>>(gpu_image1, gpu_hist1);
            gpu_image_to_histogram<<<1, 1024>>>(gpu_image2, gpu_hist2);
            gpu_histogram_distance<<<1, 256>>>(gpu_hist1, gpu_hist2, gpu_hist_distance);
            hipMemcpy(&cpu_hist_distance, gpu_hist_distance, sizeof(double), hipMemcpyDeviceToHost);
            total_distance += cpu_hist_distance;
        }
        CUDA_CHECK(hipDeviceSynchronize());
        t_finish = get_time_msec();
        printf("average distance between images %f\n", total_distance / NREQUESTS);
        printf("throughput = %lf (req/sec)\n", NREQUESTS / (t_finish - t_start) * 1e+3);
    } while (0);

    /* now for the client-server part */
    printf("\n=== Client-Server ===\n");
    total_distance = 0;
    double *req_t_start = (double *) malloc(NREQUESTS * sizeof(double));
    memset(req_t_start, 0, NREQUESTS * sizeof(double));

    double *req_t_end = (double *) malloc(NREQUESTS * sizeof(double));
    memset(req_t_end, 0, NREQUESTS * sizeof(double));

    struct rate_limit_t rate_limit;
    rate_limit_init(&rate_limit, load, 0);

    /* TODO allocate / initialize memory, streams, etc... */
    //cudaEvent_t
    work_element streams[N_STREMS];



    double ti = get_time_msec();
    if (mode == PROGRAM_MODE_STREAMS) {
        for (int i = 0; i < NREQUESTS; i++) {

            /* TODO query (don't block) streams for any completed requests.
               update req_t_end of completed requests
               update total_distance */
        	check_completed_no_block(streams);//TODO impl
            rate_limit_wait(&rate_limit);
            req_t_start[i]=get_time_msec();

            int img_idx = i % N_IMG_PAIRS;
            work_element& free_stream=find_free_stream(streams);
            free_stream.update(&images1[img_idx * IMG_DIMENSION * IMG_DIMENSION],&images2[img_idx * IMG_DIMENSION * IMG_DIMENSION],&req_t_start[i],&req_t_end[i]);
            free_stream.do_kernel();
        }
        check_completed_block(streams);
        total_distance=work_element::total_distance;


    } else if (mode == PROGRAM_MODE_QUEUE) {
    	//calc num of thread blocks that can currently run in the GPU
    	int num_of_threadblocks = calcNumOfThreadblocks(); //TODO
    	int finished=0;
    	uchar *gpu_image1, *gpu_image2;
    	CUDA_CHECK(hipMalloc(&gpu_image1,SQR(IMG_DIMENSION)*N_IMG_PAIRS));
    	CUDA_CHECK(hipMalloc(&gpu_image2,SQR(IMG_DIMENSION)*N_IMG_PAIRS));
        CUDA_CHECK(hipMemcpy(gpu_image1,images1, SQR(IMG_DIMENSION)*N_IMG_PAIRS,hipMemcpyHostToDevice));
    	CUDA_CHECK(hipMemcpy(gpu_image2,images2, SQR(IMG_DIMENSION)*N_IMG_PAIRS,hipMemcpyHostToDevice));


    	QP **cpuQPs,**gpuQPs;
    	CUDA_CHECK( hipHostAlloc(&cpuQPs, num_of_threadblocks*sizeof(QP*), 0) );
    	CUDA_CHECK( hipHostGetDevicePointer( &gpuQPs,cpuQPs ,0 ) );
    	//return 0;

    	for(int i=0;i<num_of_threadblocks;i++)
    	{
    		CUDA_CHECK( hipHostAlloc(&cpuQPs[i], sizeof(QP), 0) );
    		cpuQPs[i]->cpugpu=cpu2gpuQueue();
    		cpuQPs[i]->gpucpu=gpu2cpuQueue();
    		CUDA_CHECK( hipHostGetDevicePointer(&gpuQPs[i],cpuQPs[i],0) );
    	}
    	kernel_queue_mode<<<num_of_threadblocks, threads_queue_mode>>>(gpuQPs,gpu_image1,gpu_image2);

    	for(int i=0;i<NREQUESTS;i++)
    		{
    			int img_idx = i % N_IMG_PAIRS;
    			checkQueueComplition(num_of_threadblocks,cpuQPs,&finished, &total_distance );
    			rate_limit_wait(&rate_limit);
    			req_t_start[i] = get_time_msec();
    			QueueProduce(num_of_threadblocks,cpuQPs,img_idx,&finished, &total_distance /*,&req_t_end[i]*/);
    		}
   		for(int i=0;i<num_of_threadblocks;i++)
   			QueueProduceBlock(i,num_of_threadblocks,cpuQPs,-1,&finished, &total_distance );
   		while(finished<NREQUESTS)
   			checkQueueComplition(num_of_threadblocks,cpuQPs,&finished, &total_distance );
   		CUDA_CHECK( hipDeviceSynchronize());
        CUDA_CHECK(hipFree(gpu_image1));
   	    CUDA_CHECK(hipFree(gpu_image2));
   		for(int i=0;i<num_of_threadblocks;i++)
   			CUDA_CHECK( hipHostFree(cpuQPs[i]) );
   		CUDA_CHECK( hipHostFree(cpuQPs) );


    } else {
        assert(0);
    }
    double tf = get_time_msec();

    double avg_latency = 0;
    for (int i = 0; i < NREQUESTS; i++) {
        avg_latency += (req_t_end[i] - req_t_start[i]);
    }
    avg_latency /= NREQUESTS;

    printf("mode = %s\n", mode == PROGRAM_MODE_STREAMS ? "streams" : "queue");
    printf("load = %lf (req/sec)\n", load);
    if (mode == PROGRAM_MODE_QUEUE) printf("threads = %d\n", threads_queue_mode);
    printf("average distance between images %f\n", total_distance / NREQUESTS);
    printf("throughput = %lf (req/sec)\n", NREQUESTS / (tf - ti) * 1e+3);
    printf("average latency = %lf (msec)\n", avg_latency);
    return 0;
}
#endif
